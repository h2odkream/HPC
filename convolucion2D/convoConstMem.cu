#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <opencv2/opencv.hpp>
#include <math.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <time.h>
#define RED 2
#define GREEN 1
#define BLUE 0
#define chanDepth 3
#define blockWidth 32
#define maskWidth 3

__constant__ char d_mask_x[maskWidth*maskWidth];
__constant__ char d_mask_y[maskWidth*maskWidth];
using namespace cv;

__host__ void checkCudaState(hipError_t& cudaState,const char *message){
  /* it will print an error message if there is */
  if(cudaState != hipSuccess) printf("%s",message);
}

__device__ uchar clamp(int value){
  /* it will clamp a value if is not between 0 - 255 range */
  if(value < 0) return 0;
  if(value > 255) return 255;
  return (uchar)value;
}

__device__ void setCoords(int w,uint by,uint bx,int& d,int& dY,int& dX,int& s,int& sY,int& sX,uint iSw){
  /* it will set up the needed coords to extract data from image
     and put it into correspond shared memory section */
  uint n = maskWidth/2;
  dY = d / iSw;
  dX = d % iSw;
  sY = by * blockWidth + dY - n;
  sX = bx * blockWidth + dX - n;
  s = sY * w + sX;
}

__global__ void sobeFilt(uchar *image,uchar *resImage,int width,int height){
  /* it will do convolution between two sobel filter masks and the loaded image */
  uint image_sWidth = blockWidth+maskWidth-1;
  __shared__ uchar image_s[blockWidth+maskWidth-1][blockWidth+maskWidth-1];
  uint by = blockIdx.y, bx = blockIdx.x;
  uint ty = threadIdx.y, tx = threadIdx.x;
  int dest = ty*blockWidth+ tx,	destY, destX, srcY,	srcX, src;
  setCoords(width,by,bx,dest,destY,destX,src,srcY,srcX,image_sWidth);
  if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width) image_s[destY][destX] = image[src];
  else image_s[destY][destX] = 0;

  dest +=  blockWidth*blockWidth;
  if(1024 <= dest && dest <= 1155){ 
    setCoords(width,by,bx,dest,destY,destX,src,srcY,srcX,image_sWidth);
    if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width) image_s[destY][destX] = image[src];
    else image_s[destY][destX] = 0;
  }
  __syncthreads();


  int PvalueY = 0, PvalueX = 0, Pvalue = 0;
  uint row = by*blockWidth + ty;
  uint col = bx*blockWidth + tx;
  if(row < height && col < width){
    for(uint i = 0; i < maskWidth; i++)
      for(uint j = 0; j < maskWidth; j++){
        PvalueY += image_s[ty + i][tx + j] * d_mask_y[i * maskWidth + j];
        PvalueX += image_s[ty + i][tx + j] * d_mask_x[i * maskWidth + j];
      }

      Pvalue = sqrt((double)(PvalueY*PvalueY) + (double)(PvalueX*PvalueX));
      resImage[row * width + col] = clamp(Pvalue);
  }
}

__global__ void grayScale(uchar *image,uchar *resImage,int rows,int cols){
  /* it will turn an image to gray scale image */
  int ti = blockIdx.y*blockDim.y+threadIdx.y;
  int tj = blockIdx.x*blockDim.x+threadIdx.x;
  if(ti < rows && tj < cols){
    int pos = (ti*cols + tj)*chanDepth;
    resImage[ti*cols + tj] = image[pos+BLUE]*0.07 + image[pos+GREEN]*0.72 + image[pos+RED]*0.21;
  }
}

__host__ void getNames(char* argv,char* imgN,char* gray,char* sob,char* fileName){
  /* it will set up the needed output files names when processing an image */
  char *name = strtok(argv,"/."), format[11] = {" data.txt"};
  char grayscale[25] = {" Luna_grayscale.jpg"}, sobel[20] = {" Luna_sobel.jpg"};
  name = strtok(NULL,"/.");
  strcpy(imgN,name); strcpy(fileName,name);  strcpy(gray,name);  strcpy(sob,name);
  strcat(gray,grayscale); strcat(sob,sobel); strcat(fileName,format);
}

int main(int argc, char** argv ){
  if(argc != 3){
    printf("usage: %s <image> <numCases>\n",argv[0]);
    return -1;
  }

  Mat image;
  image = imread(argv[1],1);
  hipError_t cudaState = hipSuccess;

  if(!image.data){
    printf("No image data \n");
    return -1;
  }

   Size imgSize = image.size();
   int imgHeight = imgSize.height, imgWidth = imgSize.width;
   int cases = atoi(argv[2]);
   char fileName[30], imgName[30], grayscale[60], sobel[60];
   getNames(argv[1],imgName,grayscale,sobel,fileName);
   FILE *data= fopen(fileName,"w+");
   fprintf(data,"img name = %s,img size = %d x %d\n",imgName,imgHeight,imgWidth);
   fprintf(data,"%s\n","Using constant memory");
   fprintf(data,"%s\n","gpu time");

  do{
    clock_t start, end;
    double usedTime = 0.0;
    /* Memory data management */
    int reqMemForRawImg = imgHeight*imgWidth*image.channels()*sizeof(uchar);
    int reqMemForProcImg = imgHeight*imgWidth*sizeof(uchar);
    uchar *h_rawImage = NULL, *h_grayScale = NULL, *h_sobelImage = NULL;
    uchar *d_rawImage = NULL, *d_grayScale = NULL, *d_sobelImage = NULL;
    char h_mask_y[] = {-1,-2,-1,0,0,0,1,2,1}, h_mask_x[] = {-1,0,1,-2,0,2,-1,0,1};
    uint maskSizeY = sizeof(h_mask_y);
    uint maskSizeX = sizeof(h_mask_x);

    h_grayScale = (uchar *)malloc(reqMemForProcImg);
    h_sobelImage = (uchar *)malloc(reqMemForProcImg);

    cudaState = hipMalloc((void**)&d_rawImage,reqMemForRawImg);
    checkCudaState(cudaState,"Unallocated memory for d_rawImage\n");
    cudaState = hipMalloc((void**)&d_grayScale,reqMemForProcImg);
    checkCudaState(cudaState,"Unallocated memory for d_grayScale\n");
    cudaState = hipMalloc((void**)&d_sobelImage,reqMemForProcImg);
    checkCudaState(cudaState,"Unallocated memory for d_sobelImage\n");

    if(d_rawImage != NULL && d_grayScale != NULL && d_sobelImage != NULL){
      /* Setting kernel properties */
      h_rawImage = image.data;
      dim3 blockSize(32,32,1);
      int reqBlocksInX = ceil((double)imgHeight/32.0);
      int reqBlocksInY = ceil((double)imgWidth/32.0);
      dim3 gridSize(reqBlocksInY,reqBlocksInX,1);

      start = clock();
      /* Transfering and processing data to obtain grayimage */
      cudaState = hipMemcpy(d_rawImage,h_rawImage,reqMemForRawImg,hipMemcpyHostToDevice);
      checkCudaState(cudaState,"Impossible copy data from h_rawImage to d_rawImage\n");
      grayScale<<<gridSize,blockSize>>>(d_rawImage,d_grayScale,imgHeight,imgWidth);
      hipDeviceSynchronize();
      /* Transfering and processing data to obtain sobel image */
      cudaState = hipMemcpyToSymbol(HIP_SYMBOL(d_mask_y),h_mask_y,maskSizeY);
      checkCudaState(cudaState,"Impossible copy data from h_mask_y to d_mask_y\n");
      cudaState = hipMemcpyToSymbol(HIP_SYMBOL(d_mask_x),h_mask_x,maskSizeX);
      checkCudaState(cudaState,"Impossible copy data from h_mask_x to d_mask_x\n");
      sobeFilt<<<gridSize,blockSize>>>(d_grayScale,d_sobelImage,imgWidth,imgHeight);
      hipDeviceSynchronize();

      /* Recovering data of grayScale image to h_grayScale */
      cudaState = hipMemcpy(h_grayScale,d_grayScale,reqMemForProcImg,hipMemcpyDeviceToHost);
      checkCudaState(cudaState,"Impossible copy data from d_grayScale to h_grayScale\n");
      /* Recovering data of sobelImage to h_sobelImage */
      cudaState = hipMemcpy(h_sobelImage,d_sobelImage,reqMemForProcImg,hipMemcpyDeviceToHost);
      checkCudaState(cudaState,"Impossible copy data from d_sobelImage to h_sobelImage\n");
      end = clock();
      usedTime = ((double)(end - start))/ CLOCKS_PER_SEC;
      fprintf(data,"%f\n",usedTime);

      /* Saving Image */
      Mat grayscaleImage, sobelImage;
      grayscaleImage.create(imgHeight,imgWidth,CV_8UC1);
      sobelImage.create(imgHeight,imgWidth,CV_8UC1);
      grayscaleImage.data = h_grayScale;
      sobelImage.data = h_sobelImage;
      imwrite(grayscale,grayscaleImage);
      imwrite(sobel,sobelImage);
    }

    /* Freeing device's memory */
    if(d_rawImage != NULL) hipFree(d_rawImage);
    if(d_grayScale != NULL) hipFree(d_grayScale);
    if(d_sobelImage != NULL) hipFree(d_sobelImage);

    /* Freeing host's memory */
    // h_rawImage is a pointer to Mat's buffer, when Mat's buffer is  destroyed
    // memory is freed
    if(h_grayScale != NULL) free(h_grayScale);
    if(h_sobelImage != NULL) free(h_sobelImage);
    cases--;
  }while(cases > 0);
  fclose(data);
}
